
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

extern "C"{

struct point_t{
    float x;
    float y;
};

__global__ void line_kernel(struct point_t *pt,
    int *ret_values,
    int size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < size){
        float x = (pt + tid)->x;
        float y = (pt + tid)->y;
        ret_values[tid] = (fabs(2 * x + 3 - y) <= 1e-2);
    }
}

__global__ void circle_kernel(struct point_t *pt,
    int *ret_values,
    int size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < size){
        float x = (pt + tid)->x;
        float y = (pt + tid)->y;
        ret_values[tid] = (fabs(x * x + y*y - 5) <= 1e-2);
    }
}

__global__ void ellipse_kernel(struct point_t *pt,
    int *ret_values,
    int size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < size){
        float x = (pt + tid)->x;
        float y = (pt + tid)->y;
        ret_values[tid] = (fabs(x * x/4 + y*y/2 - 5) <= 1e-2);
    }
}

__global__ void parabola_kernel(struct point_t *pt,
    int *ret_values,
    int size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < size){
        float x = (pt + tid)->x;
        float y = (pt + tid)->y;
        ret_values[tid] = (fabs(x * 4 - y*y) <= 1e-2);
    }
}

__global__ void hyperbola_kernel(struct point_t *pt,
    int *ret_values,
    int size){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < size){
        float x = (pt + tid)->x;
        float y = (pt + tid)->y;
        ret_values[tid] = (fabs(x * x/4 + y*y/2 - 1) <= 1e-2);
    }
}


int *ptr(char *name,
    int xs_len,
    int ys_len,
    float *xs,
    float *ys,
    float delta){
        printf("%s\n", name);
    // printf("xs_len = %d, ys_len=%d, delta=%f\n", xs_len, ys_len, delta);
    struct point_t *p_managed;
    int *ret_values_managed;
    size_t size = xs_len * ys_len;
    hipMallocManaged(&p_managed, sizeof(point_t) * size);
    hipMallocManaged(&ret_values_managed, sizeof(int) * size);
    for(int i = 0; i<xs_len; i++){
        for(int j = 0; j<ys_len; j++){
            int offset = i * ys_len + j;
            (p_managed + offset)->x = xs[i];
            (p_managed + offset)->y = ys[j];
            ret_values_managed[offset] = 0;
        }
    }
    int block_size = 256;
    int num_blocks = size / block_size + 1;

    if(strcmp(name, "line") == 0)
        line_kernel<<<num_blocks, block_size>>>(p_managed, ret_values_managed, size);
    else if(strcmp(name, "circle") == 0)
        circle_kernel<<<num_blocks, block_size>>>(p_managed, ret_values_managed, size);
    else if(strcmp(name, "ellipse") == 0)
        ellipse_kernel<<<num_blocks, block_size>>>(p_managed, ret_values_managed, size);
    else if(strcmp(name, "parabola") == 0)
        parabola_kernel<<<num_blocks, block_size>>>(p_managed, ret_values_managed, size);
    else if(strcmp(name, "hyperbola") == 0)
        hyperbola_kernel<<<num_blocks, block_size>>>(p_managed, ret_values_managed, size);
    else
        printf("todo: %s\n", name);

    hipDeviceSynchronize();

    return ret_values_managed;
}

}

